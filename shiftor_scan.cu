
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <bitset>
#include <ctime>

using namespace std;

// #define DEBUG

#define WORD 32
#define ALPHABET_INITIAL ' '
#define ALPHABET_FINAL '~'
#define ASIZE (int) (ALPHABET_FINAL - ALPHABET_INITIAL + 1)

#define THREADS_PER_BLOCK 1024
#define MAX_P_LEN 32

__device__ unsigned int dagger1(unsigned int u1, unsigned int x1, unsigned int u2, unsigned int x2)
{
	return u1 + u2;
}

__device__ unsigned int dagger2(unsigned int u1, unsigned int x1, unsigned int u2, unsigned int x2)
{
	return (x1 << u2) | x2;
}

__global__ void shiftOR_GPU(unsigned int *convText, int t_len, unsigned int *convPattern, int p_len)
{
	unsigned int src_index = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ unsigned int shared_convPattern[MAX_P_LEN];
	__shared__ int AF[THREADS_PER_BLOCK];
	__shared__ int AS[THREADS_PER_BLOCK];

	if(threadIdx.x < p_len)
		shared_convPattern[threadIdx.x] = convPattern[threadIdx.x];

	__syncthreads();

	// printf("Hi. Pattern loaded.\n");

	// if(src_index > t_len)
		// printf("Thread: %d %d %d\n", threadIdx.x, src_index, t_len);

	AF[threadIdx.x] = 1;
	AS[threadIdx.x] = 0;

	for (int i = 0; i < p_len; ++i)
	{
		if(convText[src_index] == shared_convPattern[i])
			AS[threadIdx.x] |= 1 << i;
	}

	AS[threadIdx.x] = ~AS[threadIdx.x];

	if(threadIdx.x == 0 && blockIdx.x == 0)
	{
		AF[threadIdx.x] = 0;
		AS[threadIdx.x] = ~0;
	}

	// if (threadIdx.x == THREADS_PER_BLOCK - 1)
	// 	data[THREADS_PER_BLOCK * 1] = d_text[src_index + 1 * THREADS_PER_BLOCK - THREADS_PER_BLOCK + 1];

	__syncthreads();


	int stride = 1;
	while(stride < THREADS_PER_BLOCK)
	{
		int index = (threadIdx.x+1)*stride*2 - 1;
		if(index < THREADS_PER_BLOCK)
		{
			unsigned int tempF = dagger1(AF[index], AS[index], AF[index-stride], AS[index-stride]);
			unsigned int tempS = dagger2(AF[index], AS[index], AF[index-stride], AS[index-stride]);

			AF[index] = tempF;
			AS[index] = tempS;
		}
		stride = stride*2;

		__syncthreads();
	}

	stride = THREADS_PER_BLOCK/4;
	while(stride > 0)
	{
		int index = (threadIdx.x+1)*stride*2 - 1;
		if(index + stride < THREADS_PER_BLOCK)
		{
			unsigned int tempF = dagger1(AF[index+stride], AS[index+stride], AF[index], AS[index]);
			unsigned int tempS = dagger2(AF[index+stride], AS[index+stride], AF[index], AS[index]);

			AF[index] = tempF;
			AS[index] = tempS;
		}
		stride = stride / 2;
		__syncthreads();
	}

	// AF[src_index] = AF[threadIdx.x];
	// AS[src_index] = AS[threadIdx.x];
	// if(threadIdx.x < t_len)
	// 	printf("%d\n", AF[threadIdx.x]);
}

unsigned int charToUInt(char c)
{
	return (unsigned int) (c - ALPHABET_INITIAL);
}

char UintToChar(unsigned int i)
{
	// return (char) (i);
	// printf("Hi\n");

	// printf("%u\n", (i + charToUInt(ALPHABET_INITIAL)));
	return (char)(i) + ALPHABET_INITIAL;
}

char bringInRange(char c)
{
	if(c > ALPHABET_FINAL || c < ALPHABET_INITIAL)
	{
		// cout << c << " " << charToUInt(c) << endl;
		return ' ';
	}
	return c;
}

int countZero(unsigned int *M, int t_len, int count)
{
	for (int i = 0; i < t_len; ++i)
	{
		if(!M[i])
			count++;
	}
	return count;
}

void mapStringToInt(char input[], unsigned int converted[], size_t length)
{
	for (int i = 0; i < length; ++i)
	{
		if(input[i] > ALPHABET_FINAL || input[i] < ALPHABET_INITIAL)
		{
			printf("Error: String contains invalid characters\n");
			exit(0);
		}
		converted[i] = charToUInt(input[i]);
	}
	return;
}

void preSO(unsigned int *pattern, int p_len, unsigned int *S)
{
	for (int i = 0; i < ASIZE; ++i)
	{
		S[i] = 0;
	}

	for (int i = 0; i < p_len; ++i)
	{
		S[pattern[i]] |= 1 << i;
	}

	for (int i = 0; i < ASIZE; ++i)
	{
		S[i] = ~S[i];
	}

	return;
}

int shiftOR(unsigned int *pattern, int p_len, unsigned int *text, int t_len)
{
	unsigned int state;
	unsigned int S[ASIZE];
	int hit;

	unsigned int ctr = 0;

	/* pre-processing */
	preSO(pattern, p_len, S);

	#ifdef DEBUG
		cout << "Pre-processing Done\n";
	#endif

	/* searching */
	state = ~0;
	hit = (1 << (p_len - 1));
	for (int i = 0; i < t_len; ++i)
	{
		state = ((state << 1) & ~0) | S[text[i]];

		#ifdef DEBUG
			// cout << bitset<32>(state) << " & ["  << UintToChar(text[i]) << "] : " << bitset<32>(S[text[i]]) << endl;
		#endif

		if(!(state & hit))
		{
			// cout << "Found at position " <<  i - p_len + 1 << endl;
			ctr++;
		}
	}
	return ctr;
}

int main(int argc, const char **argv)
{
	#ifndef DEBUG
		if(argc != 3)
		{
			printf("Usage: %s <path/to/text/file> <path/to/pattern/file>\n", argv[0]);
			exit(0);
		}
	#endif

	#ifndef DEBUG
		FILE *t_fp = fopen(argv[1],"r");
	#else
		FILE *t_fp = fopen("data/t_s.txt", "r");
	#endif
	if (!t_fp)
	{
		printf("Unable to open text file.\n");
		exit(0);
	}

	#ifndef DEBUG
		FILE *p_fp = fopen(argv[2],"r");
	#else
		FILE *p_fp = fopen("data/p_sample.txt", "r");
	#endif
	if (!p_fp)
	{
		printf("Unable to open pattern file.\n");
		exit(0);
	}

	size_t t_len = 0, p_len = 0;
	while (getc(t_fp) != EOF)
	{
		t_len++;
	}
	rewind(t_fp);

	while (getc(p_fp) != EOF)
	{
		p_len++;
	}
	rewind(p_fp);

	t_len -= 1;
	p_len -= 1;

	// cout << p_len << " " << t_len<< endl;

	char *text = (char *) malloc(t_len);
	char *pattern = (char *) malloc(p_len);

	for (int l = 0; l < p_len; l++)
	{
		pattern[l] = bringInRange(getc(p_fp));
	}

	for (int l = 0; l < t_len; l++)
	{
		text[l] = bringInRange(getc(t_fp));
	}

	fclose(t_fp);
	fclose(p_fp);

	// cout << t_len << endl;
	// cout << text << endl;
	// cout << p_len << endl;
	// cout << pattern << endl;

	unsigned int* M = new unsigned int[t_len];
	unsigned int* AF = new unsigned int[t_len];
	unsigned int* AS = new unsigned int[t_len];

	unsigned int* convText = new unsigned int[t_len];
	mapStringToInt(text, convText, t_len);

	unsigned int convPattern[p_len];
	mapStringToInt(pattern, convPattern, p_len);

	free(text);
	free(pattern);

	if(p_len > WORD)
	{
		perror("Error: Use pattern length <= word size");
		return 0;
	}

	/****** CPU Execution ********/
	const clock_t begin_time = clock();
	int count = shiftOR(convPattern, p_len, convText, t_len);
	float runTime = (float)( clock() - begin_time ) /  CLOCKS_PER_SEC;

	printf("CPU found %d matches\n", count);
	printf("CPU Time for matching keywords: %fms\n\n", runTime*1000);


	/****** GPU Execution ********/
	// unsigned int* d_M;
	// unsigned int* d_AF;
	// unsigned int* d_AS;
	unsigned int* d_convText;
	unsigned int* d_convPattern;

	// cudaMalloc(&d_M, t_len * sizeof(unsigned int));
	// cudaMalloc(&d_AF, t_len * sizeof(unsigned int));
	// cudaMalloc(&d_AS, t_len * sizeof(unsigned int));

	hipMalloc(&d_convText, t_len * sizeof(unsigned int));
	hipMalloc(&d_convPattern, p_len * sizeof(unsigned int));

	hipEvent_t start, stop;
	hipEvent_t start_small, stop_small;
	float elapsedTime, elapsedTime_small;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	hipMemcpy(d_convText, convText, t_len * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_convPattern, convPattern, p_len * sizeof(unsigned int), hipMemcpyHostToDevice);

	hipEventCreate(&start_small);
	hipEventCreate(&stop_small);
	hipEventRecord(start_small,0);

	shiftOR_GPU <<<(t_len/THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(d_convText, t_len, d_convPattern, p_len);

	hipEventRecord(stop_small,0);
	hipEventSynchronize(stop_small);
	hipEventElapsedTime(&elapsedTime_small, start_small,stop_small);

	// cudaMemcpy(M, d_convText, t_len * sizeof(unsigned int), cudaMemcpyDeviceToHost);
	// countZero(M, t_len, count);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);

	printf("GPU found %d matches	\n", count);
	printf("GPU Kernel Time for matching keywords: %fms\n", elapsedTime_small);
	printf("GPU Total Time for matching keywords: %fms\n", elapsedTime);


	delete [] convText;
	delete [] M;
	delete [] AF;
	delete [] AS;

	hipFree(d_convText);
	hipFree(d_convPattern);
	// cudaFree(d_M);
	// cudaFree(d_AF);
	// cudaFree(d_AS);

	return 0;
}
