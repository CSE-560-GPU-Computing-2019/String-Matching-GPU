
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <bitset>
#include <ctime>

using namespace std;

#define DEBUG

#define WORD 32
#define ALPHABET_INITIAL ' '
#define ALPHABET_FINAL '~'
#define ASIZE (int) (ALPHABET_FINAL - ALPHABET_INITIAL + 1)

#define THREADS_PER_BLOCK 1024
#define MAX_P_LEN 32
#define streamcount 5
#define MAXK 1

__device__ unsigned int dagger1(unsigned int u1, unsigned int x1, unsigned int u2, unsigned int x2)
{
	return u1 + u2;
}

__device__ unsigned int dagger2(unsigned int u1, unsigned int x1, unsigned int u2, unsigned int x2)
{
	return (x1 << u2) | x2;
}

__device__ unsigned int dagger3(unsigned int x1, unsigned int x2)
{
	return x1 | x2;
}

__device__ unsigned int dagger4(unsigned int x1, unsigned int y1, unsigned int x2, unsigned int y2)
{
	return (y1 & ~x2) | y2;
}

__global__ void wuManber_GPU(unsigned int *convText, int t_len, unsigned int *convPattern, int p_len, int k, unsigned int *d_AF, unsigned int *d_AS, unsigned int *d_AW, int pos)
{
	__shared__ unsigned int shared_convPattern[MAX_P_LEN];
	__shared__ unsigned int AF[MAXK+1][THREADS_PER_BLOCK];
	__shared__ unsigned int AS[MAXK+1][THREADS_PER_BLOCK];
	__shared__ unsigned int AW[MAXK+1][THREADS_PER_BLOCK];

	unsigned int src_index = blockIdx.x * blockDim.x + threadIdx.x;

	if(threadIdx.x < p_len)
		shared_convPattern[threadIdx.x] = convPattern[threadIdx.x];

	int tx = threadIdx.x;

	//shiftOR algorithm 1

	AF[0][threadIdx.x] = 1;
	AS[0][threadIdx.x] = 0;

	for (int i = 0; i < p_len; ++i)
	{
		if(convText[src_index] == shared_convPattern[i])
			AS[0][threadIdx.x] |= 1 << i;
	}

	AS[0][threadIdx.x] = ~AS[0][threadIdx.x];

	if(threadIdx.x == 0 && blockIdx.x == 0)
	{
		AF[0][threadIdx.x] = 0;
		AS[0][threadIdx.x] = ~0;
	}

	__syncthreads();


	int stride = 1;
	while(stride < THREADS_PER_BLOCK)
	{
		int index = (threadIdx.x+1)*stride*2 - 1;
		if(index < THREADS_PER_BLOCK)
		{
			unsigned int tempF = dagger1(AF[0][index], AS[0][index], AF[0][index-stride], AS[0][index-stride]);
			unsigned int tempS = dagger2(AF[0][index], AS[0][index], AF[0][index-stride], AS[0][index-stride]);

			AF[0][index] = tempF;
			AS[0][index] = tempS;
		}
		stride = stride*2;

		__syncthreads();
	}

	stride = THREADS_PER_BLOCK/4;
	while(stride > 0)
	{
		int index = (threadIdx.x+1)*stride*2 - 1;
		if(index + stride < THREADS_PER_BLOCK)
		{
			unsigned int tempF = dagger1(AF[0][index+stride], AS[0][index+stride], AF[0][index], AS[0][index]);
			unsigned int tempS = dagger2(AF[0][index+stride], AS[0][index+stride], AF[0][index], AS[0][index]);

			AF[0][index] = tempF;
			AS[0][index] = tempS;
		}
		stride = stride / 2;
		__syncthreads();
	}

	unsigned int tempN = 0;

	for(int i = 1; i <= k; i++)
	{
		tempN = AF[i-1][tx-1] & (AF[i-1][tx-1] << 1) & (AF[i-1][tx] << 1);

		AW[i][0] = 0 << p_len;
		AS[i][0] = 1 << (p_len - k);
		AF[i][0] = 0;

		if(tx != 0)
		{
			AW[i][tx] = ~tempN;
			AS[i][tx] &= tempN;
			AF[i][tx] = 1;
		}

		__syncthreads();

		int stride = 1;
		while(stride < THREADS_PER_BLOCK)
		{
			int index = (threadIdx.x+1)*stride*2 - 1;
			if(index < THREADS_PER_BLOCK)
			{
				unsigned int tempF = dagger1(AF[i][index], AS[i][index], AF[i][index-stride], AS[i][index-stride]);
				unsigned int tempW = dagger3(AW[i][index] << AF[i][index-stride], AW[i][index-stride]);
				unsigned int tempS = dagger4(AW[i][index] << AF[i][index-stride], AS[i][index] << AF[i][index-stride], AW[i][index-stride], AS[i][index-stride]);

				AF[i][index] = tempF;
				AS[i][index] = tempS;
				AW[i][index] = tempW;
			}
			stride = stride*2;

			__syncthreads();
		}

		stride = THREADS_PER_BLOCK/4;
		while(stride > 0)
		{
			int index = (threadIdx.x+1)*stride*2 - 1;
			if(index + stride < THREADS_PER_BLOCK)
			{
				unsigned int tempF = dagger1(AF[i][index+stride], AS[i][index+stride], AF[i][index], AS[i][index]);
				unsigned int tempW = dagger3(AW[i][index+stride] << AF[i][index], AW[i][index]);
				unsigned int tempS = dagger4(AW[i][index+stride] << AF[i][index], AS[i][index+stride] << AF[i][index], AW[i][index], AS[i][index]);

				AF[i][index] = tempF;
				AS[i][index] = tempS;
				AW[i][index] = tempW;
			}
			stride = stride / 2;
			__syncthreads();
		}

		d_AF[i*THREADS_PER_BLOCK + pos * t_len + src_index] = AF[i][threadIdx.x];
		d_AS[i*THREADS_PER_BLOCK + pos * t_len + src_index] = AS[i][threadIdx.x];				//sadasdasdasd check
		d_AW[i*THREADS_PER_BLOCK + pos * t_len + src_index] = AF[i][threadIdx.x];
	}
}

__global__ void wuManber_halo_GPU(int t_len, int k, unsigned int *d_AF, unsigned int *d_AS, unsigned int *d_AW, unsigned int *R, int pos)
{
	__shared__ unsigned int AF[MAXK+1][THREADS_PER_BLOCK];
	__shared__ unsigned int AS[MAXK+1][THREADS_PER_BLOCK];
	__shared__ unsigned int AW[MAXK+1][THREADS_PER_BLOCK];

	unsigned int src_index = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	int store = (pos)*t_len - 1;

	__syncthreads();

	for(int i = 1; i <= k; i++)
	{
		unsigned int tempF = dagger1(d_AF[i*THREADS_PER_BLOCK + store], d_AS[i*THREADS_PER_BLOCK + tx], d_AF[i*THREADS_PER_BLOCK + tx], d_AS[i*THREADS_PER_BLOCK + tx]);
		unsigned int tempW = dagger3(d_AW[i*THREADS_PER_BLOCK + store] << d_AF[i*THREADS_PER_BLOCK + tx], d_AW[i*THREADS_PER_BLOCK + tx]);
		unsigned int tempS = dagger4(d_AW[i*THREADS_PER_BLOCK + store] << d_AF[i*THREADS_PER_BLOCK + tx], d_AS[i*THREADS_PER_BLOCK + store] << d_AF[i*THREADS_PER_BLOCK + tx], d_AW[i*THREADS_PER_BLOCK + tx], d_AS[i*THREADS_PER_BLOCK + tx]);

		AF[i][tx] = tempF;
		AS[i][tx] = tempS;
		AW[i][tx] = tempW;

		__syncthreads();
	}

	//dsfdsfdsfsdf sd fsd fdsff
}

unsigned int charToUInt(char c)
{
	return (unsigned int) (c - ALPHABET_INITIAL);
}

char UintToChar(unsigned int i)
{
	// return (char) (i);
	// printf("Hi\n");

	// printf("%u\n", (i + charToUInt(ALPHABET_INITIAL)));
	return (char)(i) + ALPHABET_INITIAL;
}

char bringInRange(char c)
{
	if(c > ALPHABET_FINAL || c < ALPHABET_INITIAL)
	{
		// cout << c << " " << charToUInt(c) << endl;
		return ' ';
	}
	return c;
}

void mapStringToInt(char input[], unsigned int converted[], size_t length)
{
	for (int i = 0; i < length; ++i)
	{
		if(input[i] > ALPHABET_FINAL || input[i] < ALPHABET_INITIAL)
		{
			printf("Error: String contains invalid characters\n");
			exit(0);
		}
		converted[i] = charToUInt(input[i]);
	}
	return;
}

int main(int argc, const char **argv)
{
	int k = 1;

	#ifndef DEBUG
		if(argc != 3)
		{
			printf("Usage: %s <path/to/text/file> <path/to/pattern/file>\n", argv[0]);
			exit(0);
		}
	#endif

	#ifndef DEBUG
		FILE *t_fp = fopen(argv[1],"r");
	#else
		FILE *t_fp = fopen("data/t_vvl.txt", "r");
	#endif
	if (!t_fp)
	{
		printf("Unable to open text file.\n");
		exit(0);
	}

	#ifndef DEBUG
		FILE *p_fp = fopen(argv[2],"r");
	#else
		FILE *p_fp = fopen("data/p_sample.txt", "r");
	#endif
	if (!p_fp)
	{
		printf("Unable to open pattern file.\n");
		exit(0);
	}

	size_t t_len = 0, p_len = 0;
	while (getc(t_fp) != EOF)
	{
		t_len++;
	}
	rewind(t_fp);

	while (getc(p_fp) != EOF)
	{
		p_len++;
	}
	rewind(p_fp);

	t_len -= 1;
	p_len -= 1;

	// cout << p_len << " " << t_len<< endl;

	char *text = (char *) malloc(t_len);
	char *pattern = (char *) malloc(p_len);

	for (int l = 0; l < p_len; l++)
	{
		pattern[l] = bringInRange(getc(p_fp));
	}

	for (int l = 0; l < t_len; l++)
	{
		text[l] = bringInRange(getc(t_fp));
	}

	fclose(t_fp);
	fclose(p_fp);

	// cout << t_len << endl;
	// cout << text << endl;
	// cout << p_len << endl;
	// cout << pattern << endl;

	unsigned int **AF = new unsigned int*[MAXK+1];
	unsigned int **AS = new unsigned int*[MAXK+1];
	unsigned int **AW = new unsigned int*[MAXK+1];

	for(int i = 0; i < MAXK+1; i++)
	{
		AF[i] = new unsigned int[t_len];
		AS[i] = new unsigned int[t_len];
		AW[i] = new unsigned int[t_len];
	}

	unsigned int* convText = new unsigned int[t_len];
	mapStringToInt(text, convText, t_len);

	unsigned int convPattern[p_len];
	mapStringToInt(pattern, convPattern, p_len);

	free(text);
	free(pattern);

	if(p_len > WORD)
	{
		perror("Error: Use pattern length <= word size");
		return 0;
	}

	/****** GPU Execution ********/
	unsigned int* d_AF;
	unsigned int* d_AS;
	unsigned int* d_AW;
	unsigned int* d_convText;
	unsigned int* d_convPattern;
	unsigned int* R;

	hipMalloc((void**)&d_AF, k * t_len * sizeof(unsigned int *));
	hipMalloc((void**)&d_AS, k * t_len * sizeof(unsigned int *));
	hipMalloc((void**)&d_AW, k * t_len * sizeof(unsigned int *));
	hipMalloc((void**)&R, k * t_len * sizeof(unsigned int));

	hipMalloc(&d_convText, t_len/streamcount * sizeof(unsigned int));
	hipMalloc(&d_convPattern, p_len * sizeof(unsigned int));

	hipEvent_t start, stop;
	hipEvent_t start_small, stop_small;
	float elapsedTime, elapsedTime_small;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	hipMemcpy(d_convPattern, convPattern, p_len * sizeof(unsigned int), hipMemcpyHostToDevice);

	hipEventCreate(&start_small);
	hipEventCreate(&stop_small);
	hipEventRecord(start_small,0);

	hipStream_t streams[streamcount + 1];

	for(int i = 1; i <= streamcount; i++)
	{
		hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
		hipMemcpyAsync(d_convText, convText + streamcount * i, t_len/streamcount * sizeof(unsigned int), hipMemcpyHostToDevice, streams[i]);
		wuManber_GPU <<<(t_len/(THREADS_PER_BLOCK * streamcount)) + 1, THREADS_PER_BLOCK, 0, streams[i]>>>(d_convText, t_len/streamcount, d_convPattern, p_len, 1, d_AF, d_AS, d_AW, i);
		hipStreamSynchronize(streams[i]);
	}

	hipStreamCreateWithFlags(&streams[streamcount], hipStreamNonBlocking);
	hipMemcpyAsync(d_convText, convText + (streamcount * streamcount), t_len%streamcount * sizeof(unsigned int), hipMemcpyHostToDevice, streams[streamcount]);
	wuManber_GPU <<<((t_len%streamcount)/(THREADS_PER_BLOCK)) + 1, THREADS_PER_BLOCK, 0, streams[streamcount]>>>(d_convText, t_len%streamcount, d_convPattern, p_len, 1, d_AF, d_AS, d_AW, streamcount);
	hipStreamSynchronize(streams[streamcount]);
	
	hipDeviceSynchronize();

	for(int i = 1; i <= streamcount; i++)
	{
		wuManber_halo_GPU <<<(t_len/(THREADS_PER_BLOCK * streamcount)) + 1, THREADS_PER_BLOCK, 0, streams[i]>>>(t_len/streamcount, 1, d_AF, d_AS, d_AW, R, i-1);
		// cudaMemcpyAsync(, ,  * sizeof(unsigned int), cudaMemcpyDeviceToHost, &streams[i][0]);
	}


	wuManber_halo_GPU <<<((t_len%streamcount)/(THREADS_PER_BLOCK)) + 1, THREADS_PER_BLOCK, 0, streams[streamcount]>>>(t_len%streamcount, 1, d_AF, d_AS, d_AW, R, streamcount);
	// cudaMemcpyAsync(AS,d_AS, k * t_len * sizeof(unsigned int), cudaMemcpyDeviceToHost, streams[streamcount]);

	hipDeviceSynchronize();

	for(int i = 1; i <= streamcount; i++)
		hipStreamDestroy(streams[i]);

	hipEventRecord(stop_small,0);
	hipEventSynchronize(stop_small);
	hipEventElapsedTime(&elapsedTime_small, start_small,stop_small);

	// cudaMemcpy(AF, d_AF, t_len * sizeof(unsigned int), cudaMemcpyDeviceToHost);
	// cudaMemcpy(AS, d_AS, t_len * sizeof(unsigned int), cudaMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);

	// printf("GPU found %d matches	\n", count);
	printf("GPU Kernel Time for matching keywords: %fms\n", elapsedTime_small);
	printf("GPU Total Time for matching keywords: %fms\n", elapsedTime);


	delete [] convText;
	delete [] AF;
	delete [] AS;
	delete [] AW;

	hipFree(d_convText);
	hipFree(d_convPattern);
	// cudaFree(d_M);
	// cudaFree(d_AF);
	// cudaFree(d_AS);

	return 0;
}
